#include "hip/hip_runtime.h"
#include "clutterBoxKernels.cuh"

#define GLM_FORCE_CXX98
#include <glm/glm.hpp>
#include <glm/mat4x4.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/matrix_inverse.hpp>

#include "nvidia/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>

__global__ void transformMeshes(glm::mat4* transformations, glm::mat4* normalMatrices, size_t* endIndices, DeviceMesh scene) {
    size_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex >= scene.vertexCount) {
        return;
    }

    unsigned int transformationIndex = 0;
    while(threadIndex > endIndices[transformationIndex]) {
        transformationIndex++;
    }

    glm::vec4 vertex;
    vertex.x = scene.vertices_x[threadIndex];
    vertex.y = scene.vertices_y[threadIndex];
    vertex.z = scene.vertices_z[threadIndex];
    vertex.w = 1.0;

    glm::vec4 normal;
    normal.x = scene.normals_x[threadIndex];
    normal.y = scene.normals_y[threadIndex];
    normal.z = scene.normals_z[threadIndex];
    normal.w = 0.0;

    glm::vec4 transformedVertex = transformations[transformationIndex] * vertex;
    glm::vec4 transformedNormal = normalMatrices[transformationIndex] * normal;

    transformedNormal = glm::normalize(transformedNormal);

    scene.vertices_x[threadIndex] = transformedVertex.x;
    scene.vertices_y[threadIndex] = transformedVertex.y;
    scene.vertices_z[threadIndex] = transformedVertex.z;

    scene.normals_x[threadIndex] = transformedNormal.x;
    scene.normals_y[threadIndex] = transformedNormal.y;
    scene.normals_z[threadIndex] = transformedNormal.z;

}

void randomlyTransformMeshes(DeviceMesh scene, float maxDistance, std::vector<DeviceMesh> device_meshList, std::default_random_engine randomGenerator) {
    std::vector<size_t> meshEndIndices(device_meshList.size());
    size_t currentEndIndex = 0;

    std::vector<glm::mat4> randomTransformations(device_meshList.size());
    std::vector<glm::mat4> randomNormalTransformations(device_meshList.size());

    for(unsigned int i = 0; i < device_meshList.size(); i++) {
        float yaw = float(randomGenerator() * 2.0 * M_PI);
        float pitch = float((randomGenerator() - 0.5) * M_PI);
        float roll = float(randomGenerator() * 2.0 * M_PI);

        float distanceX = maxDistance * randomGenerator();
        float distanceY = maxDistance * randomGenerator();
        float distanceZ = maxDistance * randomGenerator();

        glm::mat4 randomRotationTransformation(1.0);
        randomRotationTransformation = glm::rotate(randomRotationTransformation, yaw,   glm::vec3(0, 0, 1));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, pitch, glm::vec3(0, 1, 0));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, roll,  glm::vec3(1, 0, 0));

        glm::mat4 randomTransformation(1.0);
        randomTransformation = glm::translate(randomTransformation, glm::vec3(distanceX, distanceY, distanceZ));
        randomTransformation = randomRotationTransformation * randomTransformation;

        randomTransformations.at(i) = randomTransformation;
        randomNormalTransformations.at(i) = randomRotationTransformation;

        currentEndIndex += device_meshList.at(i).vertexCount;
        meshEndIndices.at(i) = currentEndIndex;
    }

    glm::mat4* device_transformations;
    size_t transformationBufferSize = device_meshList.size() * sizeof(glm::mat4);
    checkCudaErrors(hipMalloc(&device_transformations, transformationBufferSize));
    checkCudaErrors(hipMemcpy(device_transformations, randomTransformations.data(), transformationBufferSize, hipMemcpyHostToDevice));

    glm::mat4* device_normalMatrices;
    checkCudaErrors(hipMalloc(&device_normalMatrices, transformationBufferSize));
    checkCudaErrors(hipMemcpy(device_normalMatrices, randomNormalTransformations.data(), transformationBufferSize, hipMemcpyHostToDevice));

    size_t* device_endIndices;
    size_t startIndexBufferSize = device_meshList.size() * sizeof(size_t);
    checkCudaErrors(hipMalloc(&device_endIndices, startIndexBufferSize));
    checkCudaErrors(hipMemcpy(device_endIndices, meshEndIndices.data(), startIndexBufferSize, hipMemcpyHostToDevice));

    const size_t blockSize = 128;
    size_t blockCount = (scene.vertexCount / blockSize) + 1;
    transformMeshes<<<blockCount, blockSize>>>(device_transformations, device_normalMatrices, device_endIndices, scene);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    hipFree(device_transformations);
    hipFree(device_endIndices);
}