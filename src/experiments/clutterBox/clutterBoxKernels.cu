#include "hip/hip_runtime.h"
#include "clutterBoxKernels.cuh"
#include <iostream>

#define GLM_FORCE_CXX98
#include <glm/glm.hpp>
#include <glm/mat4x4.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/matrix_inverse.hpp>

#include <hip/hip_runtime.h>
#include <spinImage/gpu/types/DeviceOrientedPoint.h>
#include <nvidia/hip/hip_runtime_api.h>

__host__ __device__ __inline__ size_t roundSizeToNearestCacheLine(size_t sizeInBytes) {
    return (sizeInBytes + 127u) & ~((size_t) 127);
}


__global__ void detectDuplicates(SpinImage::gpu::Mesh mesh, bool* isDuplicate, SpinImage::gpu::Mesh* device_meshArray, int meshCount) {
    size_t vertexIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(vertexIndex >= mesh.vertexCount) {
        return;
    }

    int meshIndex = 0;
    size_t currentMeshBaseIndex = 0;
    size_t currentMeshVertexCount = device_meshArray[0].vertexCount;

    while(currentMeshBaseIndex + currentMeshVertexCount < vertexIndex) {
        currentMeshBaseIndex += currentMeshVertexCount;
        meshIndex++;
        currentMeshVertexCount = device_meshArray[meshIndex].vertexCount;
    }

    float3 vertex = make_float3(
            mesh.vertices_x[vertexIndex],
            mesh.vertices_y[vertexIndex],
            mesh.vertices_z[vertexIndex]);
    float3 normal = make_float3(
            mesh.normals_x[vertexIndex],
            mesh.normals_y[vertexIndex],
            mesh.normals_z[vertexIndex]);

    for(size_t i = currentMeshBaseIndex; i < vertexIndex; i++) {
        float3 otherVertex = make_float3(
                mesh.vertices_x[i],
                mesh.vertices_y[i],
                mesh.vertices_z[i]);
        float3 otherNormal = make_float3(
                mesh.normals_x[i],
                mesh.normals_y[i],
                mesh.normals_z[i]);

        // We're looking for exact matches here. Given that vertex duplications should
        // yield equivalent vertex coordinates, testing floating point numbers for
        // exact equivalence is warranted.
        if( vertex.x == otherVertex.x &&
            vertex.y == otherVertex.y &&
            vertex.z == otherVertex.z &&
            normal.x == otherNormal.x &&
            normal.y == otherNormal.y &&
            normal.z == otherNormal.z) {

            isDuplicate[vertexIndex] = true;
            return;
        }
    }

    isDuplicate[vertexIndex] = false;
}

__global__ void computeTargetIndices(SpinImage::array<signed long long> targetIndices, bool* duplicateVertices, size_t vertexCount) {
    size_t vertexIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(vertexIndex >= vertexCount) {
        return;
    }

    // The value of -1 indicates that the vertex is a duplicate of another one
    // and should therefore be discarded
    signed long long targetIndex = -1;

    bool isDuplicate = duplicateVertices[vertexIndex];

    if(!isDuplicate) {
        for(size_t i = 0; i < vertexIndex; i++) {
            // If it is a duplicate, it will get removed
            // Otherwise, it'll be added in front of the current entry
            targetIndex += duplicateVertices[i] ? 0 : 1;
        }
    }

    targetIndices.content[vertexIndex] = targetIndex;
}

SpinImage::array<signed long long> computeUniqueIndexMapping(SpinImage::gpu::Mesh boxScene, std::vector<SpinImage::gpu::Mesh> deviceMeshes, std::vector<size_t> *uniqueVertexCounts, size_t &totalUniqueVertexCount) {
    size_t sceneVertexCount = boxScene.vertexCount;
    DeviceMesh* device_meshArray;
    hipMalloc(&device_meshArray, sizeof(DeviceMesh) * deviceMeshes.size());
    hipMemcpy(device_meshArray, deviceMeshes.data(), sizeof(DeviceMesh) * deviceMeshes.size(), hipMemcpyHostToHost);

    bool* device_duplicateVertices;
    checkCudaErrors(hipMalloc(&device_duplicateVertices, sceneVertexCount * sizeof(bool)));
    detectDuplicates<<<(boxScene.vertexCount / 256) + 1, 256>>>(boxScene, device_duplicateVertices, device_meshArray, deviceMeshes.size());
    checkCudaErrors(hipDeviceSynchronize());

    bool* temp_duplicateVertices = new bool[sceneVertexCount];
    checkCudaErrors(hipMemcpy(temp_duplicateVertices, device_duplicateVertices, boxScene.vertexCount * sizeof(bool), hipMemcpyDeviceToHost));

    hipFree(device_meshArray);

    size_t baseIndex = 0;
    totalUniqueVertexCount = 0;
    for(auto mesh : deviceMeshes) {
        size_t meshUniqueVertexCount = 0;
        for(size_t i = 0; i < mesh.vertexCount; i++) {
            // Check if the vertex is unique
            if(temp_duplicateVertices[baseIndex + i] == false) {
                totalUniqueVertexCount++;
                meshUniqueVertexCount++;
            }
        }
        baseIndex += meshUniqueVertexCount;
        uniqueVertexCounts->push_back(meshUniqueVertexCount);
    }

    delete[] temp_duplicateVertices;

    SpinImage::array<signed long long> device_uniqueIndexMapping;
    device_uniqueIndexMapping.length = boxScene.vertexCount;
    checkCudaErrors(hipMalloc(&device_uniqueIndexMapping.content, boxScene.vertexCount * sizeof(signed long long)));
    computeTargetIndices<<<(boxScene.vertexCount / 256) + 1, 256>>>(device_uniqueIndexMapping, device_duplicateVertices, boxScene.vertexCount);
    checkCudaErrors(hipDeviceSynchronize());
    checkCudaErrors(hipFree(device_duplicateVertices));

    return device_uniqueIndexMapping;
}

__global__ void mapVertices(SpinImage::gpu::Mesh boxScene, SpinImage::array<SpinImage::gpu::DeviceOrientedPoint> origins, SpinImage::array<signed long long> mapping) {
    size_t vertexIndex = blockIdx.x * blockDim.x + threadIdx.x;
    if(vertexIndex >= boxScene.vertexCount) {
        return;
    }

    signed long long targetIndex = mapping.content[vertexIndex];

    if(targetIndex != -1 && targetIndex < origins.length) {
        float3 vertex = make_float3(
                boxScene.vertices_x[vertexIndex],
                boxScene.vertices_y[vertexIndex],
                boxScene.vertices_z[vertexIndex]);
        float3 normal = make_float3(
                boxScene.normals_x[vertexIndex],
                boxScene.normals_y[vertexIndex],
                boxScene.normals_z[vertexIndex]);

        SpinImage::gpu::DeviceOrientedPoint origin;
        origin.vertex = vertex;
        origin.normal = normal;

        origins.content[targetIndex] = origin;
    }
}

SpinImage::array<SpinImage::gpu::DeviceOrientedPoint> applyUniqueMapping(SpinImage::gpu::Mesh boxScene, SpinImage::array<signed long long> device_mapping, size_t totalUniqueVertexCount) {
    assert(boxScene.vertexCount == device_mapping.length);

    SpinImage::array<SpinImage::gpu::DeviceOrientedPoint> device_origins;
    device_origins.length = totalUniqueVertexCount;
    checkCudaErrors(hipMalloc(&device_origins.content, totalUniqueVertexCount * sizeof(SpinImage::gpu::DeviceOrientedPoint)));

    mapVertices<<<(boxScene.vertexCount / 256) + 1, 256>>>(boxScene, device_origins, device_mapping);
    checkCudaErrors(hipDeviceSynchronize());

    return device_origins;
}

SpinImage::array<SpinImage::gpu::DeviceOrientedPoint> computeUniqueSpinOrigins(SpinImage::gpu::Mesh &mesh) {
    std::vector<SpinImage::gpu::Mesh> deviceMeshes;
    deviceMeshes.push_back(mesh);
    std::vector<size_t> vertexCounts;
    size_t totalUniqueVertexCount;
    SpinImage::array<signed long long> device_mapping = computeUniqueIndexMapping(mesh, deviceMeshes, &vertexCounts, totalUniqueVertexCount);
    SpinImage::array<SpinImage::gpu::DeviceOrientedPoint> device_origins = applyUniqueMapping(mesh, device_mapping, totalUniqueVertexCount);
    checkCudaErrors(hipFree(device_mapping.content));
    return device_origins;
}

__global__ void transformMeshes(glm::mat4* transformations, glm::mat3* normalMatrices, size_t* endIndices, SpinImage::gpu::Mesh scene) {
    size_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex >= scene.vertexCount) {
        return;
    }

    unsigned int transformationIndex = 0;
    while(threadIndex >= endIndices[transformationIndex]) {
        transformationIndex++;
    }

    glm::vec4 vertex;
    vertex.x = scene.vertices_x[threadIndex];
    vertex.y = scene.vertices_y[threadIndex];
    vertex.z = scene.vertices_z[threadIndex];
    vertex.w = 1.0;

    glm::vec3 normal;
    normal.x = scene.normals_x[threadIndex];
    normal.y = scene.normals_y[threadIndex];
    normal.z = scene.normals_z[threadIndex];

    glm::vec4 transformedVertex = transformations[transformationIndex] * vertex;
    glm::vec3 transformedNormal = normalMatrices[transformationIndex] * normal;

    transformedNormal = glm::normalize(transformedNormal);

    scene.vertices_x[threadIndex] = transformedVertex.x;
    scene.vertices_y[threadIndex] = transformedVertex.y;
    scene.vertices_z[threadIndex] = transformedVertex.z;

    scene.normals_x[threadIndex] = transformedNormal.x;
    scene.normals_y[threadIndex] = transformedNormal.y;
    scene.normals_z[threadIndex] = transformedNormal.z;

}

void randomlyTransformMeshes(SpinImage::gpu::Mesh scene, std::vector<SpinImage::gpu::Mesh> device_meshList, std::vector<Transformation> transformations) {
    std::vector<size_t> meshEndIndices(device_meshList.size());
    size_t currentEndIndex = 0;

    std::vector<glm::mat4> randomTransformations(device_meshList.size());
    std::vector<glm::mat3> randomNormalTransformations(device_meshList.size());

    for(unsigned int i = 0; i < device_meshList.size(); i++) {
        float yaw = transformations.at(i).rotation.y;
        float pitch = transformations.at(i).rotation.x;
        float roll = transformations.at(i).rotation.z;

        float distanceX = transformations.at(i).position.x;
        float distanceY = transformations.at(i).position.y;
        float distanceZ = transformations.at(i).position.z;

        std::cout << "\t\tRotation: (" << yaw << ", " << pitch << ", "<< roll << "), Translation: (" << distanceX << ", "<< distanceY << ", "<< distanceZ << "), Vertex Count: " << device_meshList.at(i).vertexCount << std::endl;

        glm::mat4 randomRotationTransformation(1.0);
        randomRotationTransformation = glm::rotate(randomRotationTransformation, yaw,   glm::vec3(0, 0, 1));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, pitch, glm::vec3(0, 1, 0));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, roll,  glm::vec3(1, 0, 0));

        glm::mat4 randomTransformation(1.0);
        randomTransformation = glm::translate(randomTransformation, glm::vec3(distanceX, distanceY, distanceZ));
        randomTransformation = randomTransformation * randomRotationTransformation;

        randomTransformations.at(i) = randomTransformation;
        randomNormalTransformations.at(i) = glm::mat3(randomRotationTransformation);

        currentEndIndex += device_meshList.at(i).vertexCount;
        meshEndIndices.at(i) = currentEndIndex;
    }

    glm::mat4* device_transformations;
    size_t transformationBufferSize = device_meshList.size() * sizeof(glm::mat4);
    checkCudaErrors(hipMalloc(&device_transformations, transformationBufferSize));
    checkCudaErrors(hipMemcpy(device_transformations, randomTransformations.data(), transformationBufferSize, hipMemcpyHostToDevice));

    glm::mat3* device_normalMatrices;
    size_t normalMatrixBufferSize = device_meshList.size() * sizeof(glm::mat3);
    checkCudaErrors(hipMalloc(&device_normalMatrices, normalMatrixBufferSize));
    checkCudaErrors(hipMemcpy(device_normalMatrices, randomNormalTransformations.data(), normalMatrixBufferSize, hipMemcpyHostToDevice));

    size_t* device_endIndices;
    size_t startIndexBufferSize = device_meshList.size() * sizeof(size_t);
    checkCudaErrors(hipMalloc(&device_endIndices, startIndexBufferSize));
    checkCudaErrors(hipMemcpy(device_endIndices, meshEndIndices.data(), startIndexBufferSize, hipMemcpyHostToDevice));

    const size_t blockSize = 128;
    size_t blockCount = (scene.vertexCount / blockSize) + 1;
    transformMeshes<<<blockCount, blockSize>>>(device_transformations, device_normalMatrices, device_endIndices, scene);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    hipFree(device_transformations);
    hipFree(device_normalMatrices);
    hipFree(device_endIndices);
}

void randomlyTransformMeshes(SpinImage::gpu::Mesh scene, float maxDistance, std::vector<SpinImage::gpu::Mesh> device_meshList, std::minstd_rand0 &randomGenerator) {
    std::uniform_real_distribution<float> distribution(0, 1);

    std::vector<Transformation> transformations;

    for(unsigned int i = 0; i < device_meshList.size(); i++) {
        Transformation trans{};

        trans.rotation.y = float(distribution(randomGenerator) * 2.0 * M_PI);
        trans.rotation.x = float((distribution(randomGenerator) - 0.5) * M_PI);
        trans.rotation.z = float(distribution(randomGenerator) * 2.0 * M_PI);

        trans.position.x = maxDistance * distribution(randomGenerator);
        trans.position.y = maxDistance * distribution(randomGenerator);
        trans.position.z = maxDistance * distribution(randomGenerator);

        transformations.push_back(trans);

    }

    randomlyTransformMeshes(scene, device_meshList, transformations);

}
