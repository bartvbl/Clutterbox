#include "hip/hip_runtime.h"
#include "clutterBoxKernels.cuh"
#include <iostream>

#define GLM_FORCE_CXX98
#include <glm/glm.hpp>
#include <glm/mat4x4.hpp>
#include <glm/gtc/matrix_transform.hpp>
#include <glm/gtc/matrix_inverse.hpp>

#include "nvidia/hip/hip_runtime_api.h"
#include "../../../../libShapeSearch/lib/nvidia-samples-common/nvidia/hip/hip_runtime_api.h"
#include <hip/hip_runtime.h>
#include <spinImage/gpu/types/DeviceOrientedPoint.h>
#include <nvidia/hip/hip_runtime_api.h>

__host__ __device__ __inline__ size_t roundSizeToNearestCacheLine(size_t sizeInBytes) {
    return (sizeInBytes + 127u) & ~((size_t) 127);
}

__global__ void removeDuplicates(DeviceMesh inputMesh, QSIMesh outMesh, size_t* totalVertexCount) {
    // Only a single warp to avoid complications related to divergence within a block
    // (syncthreads may hang indefinitely if some threads diverged)
    const int threadCount = 32;

    // Kernel is made for a single block of threads for easy implementation
    assert(gridDim.x == 1 && gridDim.y == 1 && gridDim.z == 1);
    assert(blockDim.x == threadCount && blockDim.y == 1 && blockDim.z == 1);

    int threadIndex = threadIdx.x;

    __shared__ size_t arrayPointer;

    arrayPointer = 0;

    for(size_t vertexIndex = threadIndex; vertexIndex < inputMesh.vertexCount; vertexIndex += threadCount) {
        float3 vertex = make_float3(
                inputMesh.vertices_x[vertexIndex],
                inputMesh.vertices_y[vertexIndex],
                inputMesh.vertices_z[vertexIndex]);
        float3 normal = make_float3(
                inputMesh.normals_x[vertexIndex],
                inputMesh.normals_y[vertexIndex],
                inputMesh.normals_z[vertexIndex]);

        bool shouldBeDiscarded = false;

        for(size_t otherIndex = 0; otherIndex < vertexIndex; otherIndex++) {
            float3 otherVertex = make_float3(
                    inputMesh.vertices_x[otherIndex],
                    inputMesh.vertices_y[otherIndex],
                    inputMesh.vertices_z[otherIndex]);
            float3 otherNormal = make_float3(
                    inputMesh.normals_x[otherIndex],
                    inputMesh.normals_y[otherIndex],
                    inputMesh.normals_z[otherIndex]);

            // We're looking for exact matches here. Given that vertex duplications should
            // yield equivalent vertex coordinates, testing floating point numbers for
            // exact equivalence is warranted.
            if( vertex.x == otherVertex.x &&
                vertex.y == otherVertex.y &&
                vertex.z == otherVertex.z &&
                normal.x == otherNormal.x &&
                normal.y == otherNormal.y &&
                normal.z == otherNormal.z) {

                shouldBeDiscarded = true;
                break;
            }
        }

        unsigned int uniqueVerticesInWarp = __ballot_sync(0xFFFFFFFF, !shouldBeDiscarded);
        unsigned int uniqueVertexCount = __popc(uniqueVerticesInWarp);

        unsigned int indicesBeforeMe = __popc(uniqueVerticesInWarp << (32 - threadIndex));
        size_t outVertexIndex = arrayPointer + indicesBeforeMe;

        const size_t blockSize = roundSizeToNearestCacheLine(outMesh.vertexCount);

        if(!shouldBeDiscarded) {
            outMesh.spinOriginsBasePointer[0 * blockSize + outVertexIndex] = vertex.x;
            outMesh.spinOriginsBasePointer[1 * blockSize + outVertexIndex] = vertex.y;
            outMesh.spinOriginsBasePointer[2 * blockSize + outVertexIndex] = vertex.z;

            outMesh.spinOriginsBasePointer[3 * blockSize + outVertexIndex] = normal.x;
            outMesh.spinOriginsBasePointer[4 * blockSize + outVertexIndex] = normal.y;
            outMesh.spinOriginsBasePointer[5 * blockSize + outVertexIndex] = normal.z;
        }

        if(threadIndex == 0) {
            arrayPointer += uniqueVertexCount;
        }
    }

    // Returning the new size
    *totalVertexCount = arrayPointer;
}

array<DeviceOrientedPoint> removeDuplicates(DeviceMesh mesh) {
    std::cout << "Removing duplicate vertices.. " << std::endl;
    size_t* device_totalVertexCount;
    checkCudaErrors(hipMalloc(&device_totalVertexCount, sizeof(size_t)));

    removeDuplicates<<<1, 32>>>(mesh, outMesh, device_totalVertexCount);
    checkCudaErrors(hipDeviceSynchronize());

    size_t totalVertexCount = 0;
    checkCudaErrors(hipMemcpy(&totalVertexCount, device_totalVertexCount, sizeof(size_t), hipMemcpyDeviceToHost));

    std::cout << "\tReduced " << mesh.vertexCount << " vertices to " << totalVertexCount << "." << std::endl;

}

__global__ void transformMeshes(glm::mat4* transformations, glm::mat3* normalMatrices, size_t* endIndices, DeviceMesh scene) {
    size_t threadIndex = blockIdx.x * blockDim.x + threadIdx.x;

    if(threadIndex >= scene.vertexCount) {
        return;
    }

    unsigned int transformationIndex = 0;
    while(threadIndex >= endIndices[transformationIndex]) {
        transformationIndex++;
    }

    glm::vec4 vertex;
    vertex.x = scene.vertices_x[threadIndex];
    vertex.y = scene.vertices_y[threadIndex];
    vertex.z = scene.vertices_z[threadIndex];
    vertex.w = 1.0;

    glm::vec3 normal;
    normal.x = scene.normals_x[threadIndex];
    normal.y = scene.normals_y[threadIndex];
    normal.z = scene.normals_z[threadIndex];

    glm::vec4 transformedVertex = transformations[transformationIndex] * vertex;
    glm::vec3 transformedNormal = normalMatrices[transformationIndex] * normal;

    transformedNormal = glm::normalize(transformedNormal);

    scene.vertices_x[threadIndex] = transformedVertex.x;
    scene.vertices_y[threadIndex] = transformedVertex.y;
    scene.vertices_z[threadIndex] = transformedVertex.z;

    scene.normals_x[threadIndex] = transformedNormal.x;
    scene.normals_y[threadIndex] = transformedNormal.y;
    scene.normals_z[threadIndex] = transformedNormal.z;

}

void randomlyTransformMeshes(DeviceMesh scene, float maxDistance, std::vector<DeviceMesh> device_meshList, std::default_random_engine &randomGenerator) {
    std::vector<size_t> meshEndIndices(device_meshList.size());
    size_t currentEndIndex = 0;

    std::vector<glm::mat4> randomTransformations(device_meshList.size());
    std::vector<glm::mat3> randomNormalTransformations(device_meshList.size());

    std::uniform_real_distribution<float> distribution(0, 1);

    for(unsigned int i = 0; i < device_meshList.size(); i++) {
        float yaw = float(distribution(randomGenerator) * 2.0 * M_PI);
        float pitch = float((distribution(randomGenerator) - 0.5) * M_PI);
        float roll = float(distribution(randomGenerator) * 2.0 * M_PI);

        float distanceX = maxDistance * distribution(randomGenerator);
        float distanceY = maxDistance * distribution(randomGenerator);
        float distanceZ = maxDistance * distribution(randomGenerator);

        std::cout << "Rotation: (" << yaw << ", " << pitch << ", "<< roll << "), Translation: (" << distanceX << ", "<< distanceY << ", "<< distanceZ << "), Vertex Count: " << device_meshList.at(i).vertexCount << std::endl;

        glm::mat4 randomRotationTransformation(1.0);
        randomRotationTransformation = glm::rotate(randomRotationTransformation, yaw,   glm::vec3(0, 0, 1));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, pitch, glm::vec3(0, 1, 0));
        randomRotationTransformation = glm::rotate(randomRotationTransformation, roll,  glm::vec3(1, 0, 0));

        glm::mat4 randomTransformation(1.0);
        randomTransformation = glm::translate(randomTransformation, glm::vec3(distanceX, distanceY, distanceZ));
        randomTransformation = randomTransformation * randomRotationTransformation;

        randomTransformations.at(i) = randomTransformation;
        randomNormalTransformations.at(i) = glm::mat3(randomRotationTransformation);

        currentEndIndex += device_meshList.at(i).vertexCount;
        meshEndIndices.at(i) = currentEndIndex;
    }

    glm::mat4* device_transformations;
    size_t transformationBufferSize = device_meshList.size() * sizeof(glm::mat4);
    checkCudaErrors(hipMalloc(&device_transformations, transformationBufferSize));
    checkCudaErrors(hipMemcpy(device_transformations, randomTransformations.data(), transformationBufferSize, hipMemcpyHostToDevice));

    glm::mat3* device_normalMatrices;
    size_t normalMatrixBufferSize = device_meshList.size() * sizeof(glm::mat3);
    checkCudaErrors(hipMalloc(&device_normalMatrices, normalMatrixBufferSize));
    checkCudaErrors(hipMemcpy(device_normalMatrices, randomNormalTransformations.data(), normalMatrixBufferSize, hipMemcpyHostToDevice));

    size_t* device_endIndices;
    size_t startIndexBufferSize = device_meshList.size() * sizeof(size_t);
    checkCudaErrors(hipMalloc(&device_endIndices, startIndexBufferSize));
    checkCudaErrors(hipMemcpy(device_endIndices, meshEndIndices.data(), startIndexBufferSize, hipMemcpyHostToDevice));

    const size_t blockSize = 128;
    size_t blockCount = (scene.vertexCount / blockSize) + 1;
    transformMeshes<<<blockCount, blockSize>>>(device_transformations, device_normalMatrices, device_endIndices, scene);

    hipDeviceSynchronize();
    checkCudaErrors(hipGetLastError());

    hipFree(device_transformations);
    hipFree(device_normalMatrices);
    hipFree(device_endIndices);
}